#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void alloutputs(int *counter) {
      //(*counter)++;
      atomicAdd(counter,1);
      __syncthreads();
      printf("%d, %d\n",threadIdx.x, *counter);
}
int main() {
    int *counter, hcounter = 0;
    hipMalloc(&counter, sizeof(int));
    hipMemcpy(counter, &hcounter, sizeof(int), hipMemcpyHostToDevice);
    alloutputs<<<1, 288>>>(counter);
    hipDeviceSynchronize();
    int *n = new int; //(int *) malloc(sizeof(int));
    hipMemcpy(n,counter,sizeof(int), hipMemcpyDeviceToHost);
    printf("%d\n",*n);
    return 0;
}

#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"
#define SIZE 8

//This is a sample generation of the input array. The assignment need not be evaluated on the same matrix.
void fill_array(int *mat, int M){
  for(int i=0; i<M; i++){
    // mat[i] = rand()%SIZE+1;
    mat[i] = M-i;
  }
}

// Print the array
void print_array(int *mat, int M){
  for(int i=0; i < M; i++){
    printf("%d ", mat[i]);
  }
  printf("\n");
}

int main(int argc, char** argv)
{
  // specify the dimensions of the input array
  // const int M = 32768;
  const int M = SIZE;
  unsigned numbytes = M * sizeof(int);

  int *in = (int *) malloc(numbytes);
  int *out = (int *) malloc(numbytes);

  fill_array(in, M);
  print_array(in, M); // printing the input matrix
  int *d_in, *d_out ;

  hipError_t err;
  hipMalloc(&d_in, numbytes);
  hipMalloc(&d_out, numbytes);
  hipMemset(d_out, 0, numbytes);
  hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

  // launching the kernel
  int numThreads = 256;
  int numBlocks = (M/numThreads)+1;
  msort<<<numBlocks, numThreads>>>(d_in, d_out, M);

  /* Print the last error encountered -- helpful for debugging */
  err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));

  hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
  //print_array(out, M);

  return 0;
}
